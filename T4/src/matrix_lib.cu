// Nome: Lucas Angel Larios Prado - 2020723
// Nome: Pedro Chamberlain Matos - 1710883

#include <stdio.h>
#include <stdlib.h>
#include <pthread.h>
#include <immintrin.h>
#include <hip/hip_runtime.h>

#include "matrix_lib.h"

#define NUM_THREADS_PER_BLOCK_LIMIT 1024
#define MAX_BLOCKS_PER_GRID_LIMIT 65535

// O valor padrão de threads por bloco é 256.
int NUM_THREADS_PER_BLOCK = 256;

// O valor limite padrão de blocos por grid é 4096.
int MAX_BLOCKS_PER_GRID = 4096;

/* 

Função: validate_matrix_contents
-------------------------------
valida se a instância de matriz é válida.

retorna: caso haja sucesso, a função retorna o valor 1. em caso de erro, a função deve retornar 0.

*/

int validate_matrix_contents(struct matrix *matrix) {
    if (matrix == NULL) {
        printf("ERROR: Matrix is undeclared (NULL).");
        return 0;
    }

    if (matrix-> height < 0 || matrix->width < 0) {
        printf("ERROR: Matrix's height or width is invalid (< 0).");
        return 0;
    }

    return 1;
}

/* 

Função: validate_matrix_operations
----------------------------------
valida se as instâncias de matriz podem ser utilizadas para fazer um produto escalar e 
se a instância de matriz usada para armazenar o resultado é compátivel com as utilizadas no produto.

retorna: caso haja sucesso, a função retorna o valor 1. em caso de erro, a função deve retornar 0.

*/

int validate_matrix_operations(struct matrix *a, struct matrix *b, struct matrix *c) {
    if (validate_matrix_contents(a) == 0  || validate_matrix_contents(b) == 0 || validate_matrix_contents(c) == 0) return 0;

    if (a->width != b->height) {
        printf("ERROR: Matrixes width and height don't match.");
        return 0;
    }

    if (a->height != c->height || b->width != c->width) {
        printf("ERROR: The resulting matrix's width and height don't match with the matrixes used in the scalar operation.");
        return 0;
    }

    return 1;
}

/* 

Função: set_grid_size
--------------------------
atualiza as variáveis globais NUM_THREADS_PER_BLOCK e
NUM_BLOCKS_PER_GRID, que definem o número de threads por
blocos e o número de blocos por grid que devem ser utilizados.

caso haja sucesso, a função retorna o valor 1.

caso algum dos parâmetros extrapole um dos valores máximos
definidos no início deste arquivo, os valores atuais devem 
ser mantidos e a função retorna o valor 0.

*/

void set_grid_size(int threads_per_block, int max_blocks_per_grid) {
    if (threads_per_block > NUM_THREADS_PER_BLOCK_LIMIT) {
        printf("ERROR: Number of threads per block exceeded value");
        return 0;
    } 
    
    if (max_blocks_per_grid > MAX_BLOCKS_PER_GRID_LIMIT) {
        printf("ERROR: Max number of blocks per grid exceeded value.");
        return 0;
    } 
    
    NUM_THREADS_PER_BLOCK = threads_per_block;
    MAX_BLOCKS_PER_GRID = max_blocks_per_grid;
    return 1;
}

/* 

Função: initialize_threads
--------------------------
inicializa as threads que serão utilizadas para efetuar 
os cálculos das funções scalar_matrix_mult e matrix_matrix_mult.

*/

void initialize_threads(void *thread_routine, void *args, int args_struct_size) {
    pthread_t threads[NUM_THREADS]; 
    pthread_attr_t thread_attr;
    void *value_ptr;

    pthread_attr_init(&thread_attr);
    pthread_attr_setdetachstate(&thread_attr, PTHREAD_CREATE_JOINABLE);

    for(int i = 0; i < NUM_THREADS; i++, args += args_struct_size) {
        pthread_create(&threads[i], &thread_attr, thread_routine, args);
        pthread_join(threads[i], &value_ptr);
    }
}

/* 

Função: scalar_matrix_mult_routine
--------------------------
rotina iniciada por uma thread para fazer parte do processo 
de cálculo do produto de um valor escalar  em uma matriz.

thread_args: parâmetros da thread. 

para mais informações sobre esses parâmetros, verifique 
a definição da struct scalar_matrix_thread_args.

*/

int scalar_matrix_mult_routine(void *thread_args) {
    struct scalar_matrix_thread_args *args = 
        (struct scalar_matrix_thread_args *) thread_args;

    float *m_curr = args->m_array_start, 
        *m_end = args->m_array_start + args->m_array_length;

    __m256 curr, result, 
        scalar = _mm256_set1_ps(args->scalar);

    for (; m_curr <= m_end; m_curr += 8) {
        curr = _mm256_load_ps(m_curr);
        result = _mm256_mul_ps(curr, scalar);
        _mm256_store_ps(m_curr, result);
    }

    pthread_exit(NULL);
}

/* 

Função: scalar_matrix_mult
--------------------------
inicia o processo de cálculo do produto de um valor escalar em uma matriz.

scalar_value: valor escalar utilizada no cálculo. 
matrix: matriz a ser utilizada no cálculo.

retorna: caso haja sucesso, a função retorna o valor 1. em caso de erro, a função deve retornar 0.

*/

int scalar_matrix_mult(float scalar_value, struct matrix *matrix) {
    float *m_curr;
    int rows_per_thread, m_array_length;
    struct scalar_matrix_thread_args args[NUM_THREADS];

    if (validate_matrix_contents(matrix) == 0) return 0;

    m_curr = matrix->rows;
    rows_per_thread = matrix->height / NUM_THREADS;
    m_array_length = rows_per_thread * matrix->width;

    for (int i = 0; i < NUM_THREADS; i++, m_curr += m_array_length) {
        args[i].m_array_start = m_curr;
        args[i].m_array_length = m_array_length;
        args[i].scalar = scalar_value;
    }

    initialize_threads(scalar_matrix_mult_routine, args, sizeof(struct scalar_matrix_thread_args));
    return 1;
}

/* 

Função: matrix_matrix_mult_routine
--------------------------
rotina iniciada por uma thread para fazer parte do processo 
de cálculo do produto entre duas matrizes A e B, 
armazenando o resultado numa matriz C.

thread_args: parâmetros da thread. 

para mais informações sobre esses parâmetros, verifique 
a definição da struct matrix_matrix_thread_args.

*/

int matrix_matrix_mult_routine(void *thread_args) {
    struct matrix_matrix_thread_args *args = 
        (struct matrix_matrix_thread_args *) thread_args;

    int a_column = 0, 
        a_row = 0;

    float *a_curr = args->a_start,
        *b_curr, *c_curr;

    __m256 matrix_a_avx, matrix_b_avx, matrix_c_avx, result_avx;

    for (; a_row < args->rows_per_thread; a_curr++) {
        matrix_a_avx = _mm256_set1_ps(*a_curr);

        b_curr = args->b_start;
        b_curr += args->b_width * a_column;

        c_curr = args->c_start;
        c_curr += args->c_width * a_row;

        for (int curr_column = 0; curr_column < args->b_width; curr_column += 8, b_curr += 8, c_curr += 8) {
            matrix_b_avx = _mm256_load_ps(b_curr);
            matrix_c_avx = _mm256_load_ps(c_curr);
            result_avx = _mm256_fmadd_ps(matrix_a_avx, matrix_b_avx, matrix_c_avx);
			_mm256_store_ps(c_curr, result_avx);
        }

        if (a_column + 1 == args->a_width) {
            a_column = 0;
            a_row++;
        } else {
            a_column++;
        }
    }

    pthread_exit(NULL);
}

/* 

Função: matrix_matrix_mult
--------------------------
inicia o processo do cálculo do produto entre duas matrizes A e B, 
armazenando o resultado numa matriz C.

a: matriz A, a ser utilizada no cálculo.
b: matriz B, a ser utilizada no cálculo.
c: matriz C, resultado armazenado do cálculo entre as matrizes A e B.

retorna: caso haja sucesso, a função retorna o valor 1. em caso de erro, a função deve retornar 0.

*/

int matrix_matrix_mult(struct matrix *a, struct matrix *b, struct matrix *c) {
    float *a_curr, *c_curr;
    int rows_per_thread, a_array_length, c_array_length;
    struct matrix_matrix_thread_args args[NUM_THREADS];

    if (validate_matrix_operations(a, b, c) == 0) return 0;

    a_curr = a->rows;
    c_curr = c->rows;
    rows_per_thread = c->height / NUM_THREADS;
    a_array_length = rows_per_thread * a->width;
    c_array_length = rows_per_thread * c->width;

    for (int i = 0; i < NUM_THREADS; i++, a_curr += a_array_length, c_curr += c_array_length) {
        args[i].a_start = a_curr;
        args[i].b_start = b->rows;
        args[i].c_start = c_curr;
        args[i].a_width = a->width;
        args[i].b_width = b->width;
        args[i].c_width = c->width;
        args[i].rows_per_thread = rows_per_thread;
    }

    initialize_threads(matrix_matrix_mult_routine, args, sizeof(struct matrix_matrix_thread_args));
    return 1;
}